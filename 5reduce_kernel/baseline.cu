#include <hip/hip_runtime.h>

#include <cstdio>


__global__ void reduce_baseline(int* input, int* output, size_t n) {

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    printf("tid: %d\n", idx);

    int sum = 0;
    for (int i = 0; i < n; ++i) {
        sum += input[i];
    }

    *output = sum;
}

bool CheckResult(int *out, int groudtruth) {
    if (*out != groudtruth) {
        return false;
    }
    return true;
}

int main() {

    // 获取设备属性
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    // thread
    int gridSize  = 1;
    int blockSize = 1;
    dim3 Grid(gridSize);
    dim3 Block(blockSize);

    // allocate memory
    int* device_in;
    int* device_out;
    const int N = 25600000;
    int* host_in  = (int* )malloc(N * sizeof(int));
    int* host_out = (int* )malloc(sizeof(int));
    int gt = 0;
    for (int i = 0; i < N; ++i) {
        host_in[i] = 1;
        gt += host_in[i];
    }
    hipMalloc((void** )&device_in, N * sizeof(int));
    hipMalloc((void** )&device_out, sizeof(int));
    hipMemcpy(device_in, host_in, N * sizeof(int), hipMemcpyHostToDevice);

    // 开始处理
    // CPU 的方式处理, 分配 1 个 thread
    float ms = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    reduce_baseline<<<Grid, Block>>>(device_in, device_out, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    hipMemcpy(host_out, device_out, 1 * sizeof(int), hipMemcpyDeviceToHost);

    // 验证
    if (CheckResult(host_out, gt)) {
        printf("the ans is right\n");
    }
    else {
        printf("groudtruth: %d\n", gt);
        printf("result: %d\n", host_out);
        printf("the ans is wrong\n");
    }
    printf("reduce_baseline latency = %f ms\n", ms);

    // free
    hipFree(device_in);
    hipFree(device_out);
    free(host_in);
    free(host_out);


    return 0;
}