// reduce 类算子 -> 累加
// v4: 最一个 warp 不进入 for 循环, 独立出来
// baseline:  519.084167 ms
// reduce_v0: 0.696928 ms      bank conflict: 834
// reduce_v1: 0.510880 ms(old) bank conflict: 7001056
// reduce_v1: 0.478528 ms(new) bank conflict: 689
// reduce_v2: 0.454912 ms      bank conflict: 891
// reduce_v3: 0.237376 ms      bank conflict: 987
// reduce_v4: 0.197024 ms      bank conflict: 2343


#include <hip/hip_runtime.h>

#include <cstdio>

#define blockSize 128


void CheckResult(int *out, int groudtruth, int n) {
    float res = 0;
    for (int i = 0; i < n; i++){
        res += out[i];
    }
    if (res != groudtruth) {
        printf("groudtruth: %d\n", groudtruth);
        printf("result: %d\n", res);
        printf("the ans is wrong\n");
    }

    else
        printf("the ans is right\n");
}

// __device__: GPU 端的函数, 由编译器自行决定是否 inline
// __device__: GPU 端的函数, 由编译器自行决定是否 inline
__device__ void WarpSharedMemReduce(volatile float* smem, int tid){

    if (blockDim.x >= 64) {
        smem[tid] += smem[tid + 32];
        __syncwarp();
    }
    smem[tid] += smem[tid + 16];
    __syncwarp();
    smem[tid] += smem[tid + 8];
    __syncwarp();
    smem[tid] += smem[tid + 4];
    __syncwarp();
    smem[tid] += smem[tid + 2];
    __syncwarp();
    smem[tid] += smem[tid + 1];
    __syncwarp();
}

__global__ void reduce_v4(int* input, int* output, size_t n) {

    __shared__ float smem[blockSize];
    unsigned int tid = threadIdx.x;
    unsigned int gtid = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
    // load
    smem[tid] = input[gtid] + input[gtid + blockSize];
    __syncthreads();
    // operation
    // 单独将最后一个 warp 拿出来 -> syncwarp, 避免 __syncthreads
    for (unsigned int idx = blockDim.x / 2; idx > 32; idx >>= 1) {
        if (tid < idx) {
            smem[tid] += smem[tid + idx];
        }
        __syncthreads();
    }
    if (tid < 32) {
        WarpSharedMemReduce(smem, tid);
    }

    if (tid == 0) {
        output[blockIdx.x] = smem[0];
    }
}


int main() {

    float ms = 0;
    int groudtruth = 0;
    const int N = 25600000; // 数据量

    // 获取设备属性
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    // thread
    int gridSize  = std::min((N + 256 - 1) / 256, deviceProp.maxGridSize[0]); // 向上加 1 -> 防止 N = 255 -> gridSize = 0

    dim3 Grid(gridSize);
    dim3 Block(128);

    // allocate memory
    int* device_in;
    int* device_out;
    int* host_in  = (int* )malloc(N * sizeof(int));
    int* host_out = (int* )malloc(gridSize * sizeof(int));
    for (int i = 0; i < N; ++i) {
        host_in[i] = 1;
        groudtruth += host_in[i];
    }
    hipMalloc((void** )&device_in, N * sizeof(int));
    hipMalloc((void** )&device_out, gridSize * sizeof(int));
    hipMemcpy(device_in, host_in, N * sizeof(int), hipMemcpyHostToDevice);

    // 开始处理
    // grid: (100000, 1, 1)   block: (256, 1, 1)   thread = 25600000
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    reduce_v4<<<Grid, Block>>>(device_in, device_out, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    hipMemcpy(host_out, device_out, gridSize * sizeof(int), hipMemcpyDeviceToHost);

    // 验证
    CheckResult(host_out, groudtruth, gridSize);

    printf("reducev4 latency = %f ms\n", ms);

    // free
    hipFree(device_in);
    hipFree(device_out);
    free(host_in);
    free(host_out);


    return 0;
}