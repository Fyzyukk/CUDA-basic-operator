// reduce 类算子 -> 累加
// v7: 最后结果是串行循环 100000 次对核函数返回的结果进行求和 -> 并行
// baseline:  519.084167 ms
// reduce_v0: 0.696928 ms      bank conflict: 834
// reduce_v1: 0.510880 ms(old) bank conflict: 7001056
// reduce_v1: 0.478528 ms(new) bank conflict: 689
// reduce_v2: 0.454912 ms      bank conflict: 891
// reduce_v3: 0.237376 ms      bank conflict: 987
// reduce_v4: 0.197024 ms      bank conflict: 2343
// reduce_v5: 0.236256 ms
// reduce_v6: 0.202784 ms(blockSize = 128)
// reduce_v7: 0.422016 ms(两次 reduce)


#include <hip/hip_runtime.h>

//#include <bits/stdc++.h>
#include <cstdio>


void CheckResult(int *out, float res, int groudtruth, int n) {

    if (res != groudtruth) {
        printf("groudtruth: %d\n", groudtruth);
        printf("result: %d\n", res);
        printf("the ans is wrong\n");
    }

    else
        printf("the ans is right\n");
}

// __device__ GPU 端的函数, 编译器自行决定是否 inline
// 将 for 循环展开, 节省位运算

template <int blockSize>
__device__ void BlockSharedMemReduce(float* smem) {

    if (blockSize >= 1024) {
        if (threadIdx.x < 512)
            smem[threadIdx.x] += smem[threadIdx.x + 512];
    }
    __syncthreads();
    if (blockSize >=512) {
        if (threadIdx.x < 256)
            smem[threadIdx.x] += smem[threadIdx.x + 256];
    }
    __syncthreads();
    if (blockSize >= 256) {
        if (threadIdx.x< 128)
            smem[threadIdx.x] += smem[threadIdx.x + 128];
    }
    __syncthreads();
    if (blockSize >= 128) {
        if (threadIdx.x < 64)
            smem[threadIdx.x] += smem[threadIdx.x + 64];
    }
    __syncthreads();

    // final warp
    // volatile: 
    if (threadIdx.x < 32) {
        volatile float* vshm = smem;
        if (blockDim.x >= 64)
            vshm[threadIdx.x] += vshm[threadIdx.x + 32];
        vshm[threadIdx.x] += vshm[threadIdx.x + 16];
        vshm[threadIdx.x] += vshm[threadIdx.x + 8];
        vshm[threadIdx.x] += vshm[threadIdx.x + 4];
        vshm[threadIdx.x] += vshm[threadIdx.x + 2];
        vshm[threadIdx.x] += vshm[threadIdx.x + 1];
    }
}

template<int blockSize>
__global__ void reduce_v7(int* input, int* output, size_t n) {

    __shared__ float smem[blockSize];
    unsigned int tid = threadIdx.x;

    // 不显式指定每个 thread 处理多少个数据
    // unsigned int gtid = blockIdx.x * blockDim.x * 2 + threadIdx.x;
    unsigned int gtid = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int total_tid = blockDim.x * gridDim.x;

    // load
    // 更灵活, 不局限于 n thread -> 2n 数据   
    // 根据 for 自动确定每个 thread 处理多少个数据
    // smem[tid] = input[gtid] + input[gtid + blockDim.x];
    float sum = 0.0f;
    for (int32_t i = gtid; i < n; i += total_tid) 
        sum += input[i];

    smem[tid] = sum;
    __syncthreads();

    //operation
    BlockSharedMemReduce<blockSize>(smem);

    // result
    if (tid == 0) {
        output[blockIdx.x] = smem[0];
    }
}

int main() {

    float ms = 0;
    int groudtruth = 0;
    const int N = 25600000; // 数据量

    // 获取设备属性
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    // thread
    const int blockSize = 256;
    const int blockSize_v3 = blockSize / 2;
    const int blockSize_v6 = blockSize / 4;
    int gridSize  = std::min((N + 256 - 1) / 256, deviceProp.maxGridSize[0]); // 向上加 1 -> 防止 N = 255 -> gridSize = 0

    dim3 Grid(gridSize);
    dim3 Block(blockSize_v3);

    // allocate memory
    int* device_in;
    int* device_out;
    int* device_res;
    int* host_in  = (int* )malloc(N * sizeof(int));
    int* host_res = (int* )malloc(1 * sizeof(int));
    int* host_out = (int* )malloc(gridSize * sizeof(int));
    for (int i = 0; i < N; ++i) {
        host_in[i] = 1;
        groudtruth += host_in[i];
    }
    hipMalloc((void** )&device_in, N * sizeof(int));
    hipMalloc((void** )&device_res, 1 * sizeof(int));
    hipMalloc((void** )&device_out, gridSize * sizeof(int));
    hipMemcpy(device_in, host_in, N * sizeof(int), hipMemcpyHostToDevice);

    // 开始处理
    // grid: (100000, 1, 1)   block: (256, 1, 1)   thread = 25600000
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    reduce_v7<blockSize_v3><<<Grid, Block>>>(device_in, device_out, N);
    reduce_v7<blockSize_v3><<<1, Block>>>(device_out, device_res, gridSize);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    hipMemcpy(host_res, device_res, 1 * sizeof(int), hipMemcpyDeviceToHost);

    // 验证
    CheckResult(host_out, *host_res, groudtruth, gridSize);
    printf("reducev7 latency = %f ms\n", ms);

    // free
    hipFree(device_in);
    hipFree(device_out);
    hipFree(device_res);

    free(host_in);
    free(host_out);
    free(host_res);

    return 0;
}