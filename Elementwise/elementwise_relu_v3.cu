// shared memory
// v0运行时间: 431.14us
// v0带宽利用率: 83.56%
// v0内存吞吐量: 616.56GB/s
// v1运行时间: 393.63us
// v1带宽利用率: 92.28%
// v1内存吞吐量: 675.51GB/s
// v2运行时间: 531.39us
// v2带宽利用率: 71.82%
// v2内存吞吐量: 500.08GB/s
// v3运行时间: 458.62us
// v3带宽利用率: 78.66%
// v3内存吞吐量: 579.75GB/s

#include <iostream>
#include <hip/hip_runtime.h>


#define N 32 * 1024 * 1024
#define BlOCK_SIZE 1024

void checkout (float *output_h, float *output_host, int n) {
    for (int i = 0; i < n; ++i) {
        if (std::abs(output_h[i] - output_host[i] > 0.0001)) {
            std::cout << "output_h[" << i << "]:    " << output_h[i] << std::endl;
            std::cout << "output_host[" << i << "]: " << output_host[i] << std::endl;
            std::cout << "FAILED" << std::endl;
            return;
        }
    }
    for (int i = 10000; i < 10010; ++i) {
        std::cout << "output_h[" << i << "]:    " << output_h[i] << std::endl;
        std::cout << "output_host[" << i << "]: " << output_host[i] << std::endl;
    }
    std::cout << "PASSED" << std::endl;
}

void relu_CPU(float *input, float *output, int n) {
    for (int i = 0; i < n; ++i) {
        output[i] = input[i] > 0 ? input[i] : 0;
    }
}

// __global__ void relu(float *input, float *output) {
//     int tid = threadIdx.x;
//     int gid = blockIdx.x * blockDim.x + threadIdx.x;
//     if (gid * 4 >= N) {
//         return;
//     }

//     // global to shared
//     __shared__ float4 smem[BlOCK_SIZE]; // 1024 * 32/8 = 4096 -> 4KB
//     if (gid < N / 4) {
//         smem[tid] = reinterpret_cast<float4*>(input)[gid];
//     }
//     __syncthreads();

//     if (gid < N / 4) {
//         output[gid * 4 + 0] = smem[tid].x > 0 ? smem[tid].x : 0;
//         output[gid * 4 + 1] = smem[tid].y > 0 ? smem[tid].y : 0;
//         output[gid * 4 + 2] = smem[tid].z > 0 ? smem[tid].z : 0;
//         output[gid * 4 + 3] = smem[tid].w > 0 ? smem[tid].w : 0;
//     }
// }

__global__ void relu(float *input, float *output) {
    int tid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= N) {
        return;
    }

    // global to shared
    __shared__ float smem[BlOCK_SIZE]; // 1024 * 32/8 = 4096 -> 4KB
    smem[tid] = input[gid];
    __syncthreads();

    output[gid] = smem[tid] > 0 ? smem[tid] : 0;
}

int main() {
    float *input_h;
    float *output_h;
    float *input_d;
    float *output_d;
    float *input_host;
    float *output_host;
    int32_t elem_cnt = N; // 32M
    input_h = (float*)malloc(elem_cnt * sizeof(float));
    output_h = (float*)malloc(elem_cnt * sizeof(float));
    input_host = (float*)malloc(elem_cnt * sizeof(float));
    output_host = (float*)malloc(elem_cnt * sizeof(float));
    hipMalloc((void**)&input_d, sizeof(float) * elem_cnt);
    hipMalloc((void**)&output_d, sizeof(float) * elem_cnt);
    for (int i = 0; i < elem_cnt; ++i) {
        input_host[i] = i - 10000;
    }
    hipMemcpy(input_d, input_host, sizeof(float) * elem_cnt, hipMemcpyHostToDevice);

    int blockSize = BlOCK_SIZE;
    int blockNum = (elem_cnt + blockSize - 1) / blockSize;
    dim3 grid(blockNum);
    dim3 block(blockSize);
    std::cout << "=========== call kernel ===========" << std::endl;
    relu<<<grid, block>>>(input_d, output_d);
    std::cout << "=========== call kernel done ===========" << std::endl;
    hipMemcpy(output_h, output_d, sizeof(float) * elem_cnt, hipMemcpyDeviceToHost);

    std::cout << "=========== verfiy ===========" << std::endl;
    relu_CPU(input_host, output_host, elem_cnt);
    checkout(output_h, output_host, elem_cnt);

    free(input_h);
    free(output_h);
    free(input_host);
    free(output_host);
    hipFree(input_d);
    hipFree(output_d);


    return 0;
}
