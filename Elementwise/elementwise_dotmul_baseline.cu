// baseline
// v0: Memory Throughput [%]	95.31
// v0: Memory Throughput [Gbyte/second]	702.51

#include <time.h>
#include <iostream>
#include <algorithm>
#include <hip/hip_runtime.h>


#define N 32 * 1024 * 1024
#define BLOCK_SIZE 512
#define DEVICE_FUNCTION __device__ __host__ __forceinline__

template<typename T>
void checkout(T *output_h, T *output_host) {
    for (int i = 0; i < N; ++i) {
        if (std::abs(output_h[i] - output_host[i]) > 0.001) {
            std::cout << "output_d[" << i << "]: " << output_h[i] << std::endl;
            std::cout << "output_host[" << i << "]: " << output_host[i] << std::endl;
            std::cout << "FAILED" << std::endl;
        }
    }
    for (int i = 0; i < 10; ++i) {
        std::cout << "output_d[" << i << "]: " << output_h[i] << std::endl;
        std::cout << "output_host[" << i << "]: " << output_host[i] << std::endl;
    }
    std::cout << "PASSED" << std::endl;
}

template<typename T>
struct MultiplyFunctor {
    DEVICE_FUNCTION T operator()(T x, T y) {
        return x * y;
    }
    DEVICE_FUNCTION T operator()(T x, T y, T z) {
        return x * y * z;
    }
};

template<typename T>
void dotmul_CPU(T *input1, T *input2, T *input3, T *output) {
    MultiplyFunctor<T> dotmul;
    for (int i = 0; i < N; ++i) {
        output[i] = dotmul(input1[i], input2[i], input3[i]);
    }
}

// 每个thread负责一个数据
template<typename T>
__global__ void dotmul(T *input1, T *input2, T *input3, T *output) {
    MultiplyFunctor<T> dotmul;
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid < N) {
        output[gid] = dotmul(input1[gid], input2[gid], input3[gid]);
    }
}

int main() {
    float *input1_host;
    float *input2_host;
    float *input3_host;
    float *output_host;
    float *input1_d;
    float *input2_d;
    float *input3_d;
    float *output_d;

    input1_host = (float*)malloc(sizeof(float) * N);
    input2_host = (float*)malloc(sizeof(float) * N);
    input3_host = (float*)malloc(sizeof(float) * N);
    output_host = (float*)malloc(sizeof(float) * N);

    hipMalloc((void**)&input1_d, sizeof(float) * N);
    hipMalloc((void**)&input2_d, sizeof(float) * N);
    hipMalloc((void**)&input3_d, sizeof(float) * N);
    hipMalloc((void**)&output_d, sizeof(float) * N);

    for (int i = 0; i < N; ++i) {
        input1_host[i] = 1.0;
        input2_host[i] = 2.0;
        input3_host[i] = 3.0;
    }

    hipMemcpy(input1_d, input1_host, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(input2_d, input2_host, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(input3_d, input3_host, sizeof(float) * N, hipMemcpyHostToDevice);

    int block_size = BLOCK_SIZE;
    int block_num = (N + block_size - 1) / block_size;
    dim3 block(block_size);
    dim3 grid(block_num);
    
    std::cout << "=========== call kernel ===========" << std::endl;
    dotmul<float><<<grid, block>>>(input1_d, input2_d, input3_d, output_d);
    hipDeviceSynchronize();
    std::cout << "=========== call kernel done ===========" << std::endl;

    std::cout << "=========== verify ===========" << std::endl;
    hipMemcpy(output_host, output_d, sizeof(float) * N, hipMemcpyDeviceToHost);
    dotmul_CPU<float>(input1_host, input2_host, input3_host, output_host);
    checkout<float>(output_host, output_host);

    free(input1_host);
    free(input2_host);
    free(output_host);
    hipFree(input1_d);
    hipFree(input2_d);
    hipFree(output_d);

    return 0;
}