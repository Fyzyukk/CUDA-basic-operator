// 针对warp divergent
// v0运行时间: 933.44us
// v0带宽利用率: 62.67%
// v0内存吞吐量: 144.33GB/s
// v1运行时间: 675.90us
// v1带宽利用率: 86.46%
// v1内存吞吐量: 199.35GB/s

#include <hip/hip_runtime.h>

#include <time.h>
#include <vector>
#include <iostream>

#define N 32 * 1024 * 1024 // 32MB
#define BLOCK_SIZE 256


void CPU_reduce(std::vector<float> &input_, double &output_) {
    for (auto x : input_) {
        output_ += x;
    }
}

bool checkout(float output_, float output_host) {
    if (std::abs(output_ - output_host) > 0.0001) {
        return false;
    } else {
        return true;
    }
}

__global__ void reduce_v1(float *g_idata, float *g_odata) {
    
    // 256 * 32/8 = 1024Byte -> 1KB
    // 3080: 单个SM的L1 cache 128KB
    __shared__ float smem[BLOCK_SIZE];
    unsigned int tid = threadIdx.x;
    unsigned int gid = blockIdx.x * blockDim.x + threadIdx.x;

    // SM, 每个block独享
    // <<<N / BLOCK_SIZE, BLOCK_SIZE(向上取整)>>>
    smem[tid] = g_idata[gid];
    __syncthreads(); // 使用smem, 同步

    // =============================== change ===============================
    for (unsigned int i = 1; i < blockDim.x; i *= 2) {
        
        // 第一次迭代: i = 1
        // 0-3号warp: tid: 0 -> 127     均进入if分支
        // 4-7号warp: tid: 128 -> 255   均未进入if分支
        // 第二次迭代: i = 2
        // 同上
        // 第四次迭代: i = 8
        // 0号warp: tid: 0 -> 15    进入if分支
        //          tid: 16 -> 31   未进入if分支 -> warp divergent
        int index = 2 * i * tid;
        if (index < blockDim.x) {
            smem[index] += smem[index + i];
        }
        __syncthreads();
    }

    if (tid == 0) {
        // 写回每个block的sum
        g_odata[blockIdx.x] = smem[0];
    }
}

int main() {
    float *input_device;
    float *output_device;
    float *input_host = (float*)malloc(N * sizeof(float));
    float *output_host = (float*)malloc(N / BLOCK_SIZE * sizeof(float));
    hipMalloc((void**)&input_device, N * sizeof(float));
    hipMalloc((void**)&output_device, (N / BLOCK_SIZE) * sizeof(float));
    for (int i = 0; i < N; ++i) {
        input_host[i] = 1.0;
    }
    hipMemcpy(input_device, input_host, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 grid(N / BLOCK_SIZE);
    dim3 block(BLOCK_SIZE);
    int block_num = (N + BLOCK_SIZE - 1) / BLOCK_SIZE; // 向上取整
    reduce_v1<<<grid, block>>>(input_device, output_device);
    hipMemcpy(output_host, output_device, block_num * sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 1; i < N / BLOCK_SIZE; ++i) {
        output_host[0] += output_host[i];
    }

    bool res;
    std::vector<float> input_(N, 1.0);
    double output_ = 0;
    CPU_reduce(input_, output_);
    res = checkout(output_, output_host[0]);
    if (res) {
        std::cout << "PASSED!" << std::endl;
    } else {
        std::cout << "FAILED!" << std::endl;
    }

    free(input_host);
    free(output_host);
    hipFree(input_device);
    hipFree(output_device);

    return 0;
}